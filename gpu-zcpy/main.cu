#include <fcntl.h>
#include <assert.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

bool VERBOSE = false;
bool MAP_HOST = true;

typedef unsigned long int uint64;

typedef struct bigInt {
    uint64* data;
    uint64* dev_data;
    int len;
} bigInt;

void panic(char* msg) {
    printf("%s\n", msg);
    exit(1);
}

uint64 mul_lo(uint64 A, uint64 B) {
    return A*B;
}

void mul_hi(uint64* dst, uint64 A, uint64 B) {
    __asm__(
            "umulh x1, x1, x2\n"
            "str x1, [x0]"
   );
}

bool has_carry(uint64 A, uint64 B) {
    bool rtn = ((A&B)|((A|B)&(~(A+B))))>>63 == 1;
    return rtn;
}

__global__ void kernal_create_table(uint64* M, bigInt* pA, bigInt* pB) {
    bigInt A = *pA;
    bigInt B = *pB;
    int i = threadIdx.x/B.len;
    int j = threadIdx.x%B.len;
    // cs.opensource.google/go/go/+/master:src/math/bits/bits.go
    // lo and hi multiplication for 64 bits
    uint64 x = A.data[i];
    uint64 y = B.data[j];
    uint64 x0 = (x<<32)>>32;
    uint64 x1 = x>>32;
    uint64 y0 = (y<<32)>>32;
    uint64 y1 = y>>32;
    uint64 w0 = x0*y0;
    uint64 t = x1*y0 + (w0>>32);
    uint64 w1 = (t<<32)>>32;
    uint64 w2 = t>>32;
    w1 += x0*y1;
    uint64 hi = x1*y1 + w2 + (w1>>32);
    uint64 lo = x*y;
    *(M+(i*B.len*2)+(j*2)+(1)) = hi;
    *(M+(i*B.len*2)+(j*2)) = lo;
}

void mul(bigInt* pDst, bigInt* pA, bigInt* pB) {
    bigInt A = *pA;
    bigInt B = *pB;
    bigInt dst = *pDst;

    // Create matrix
    uint64* M;
    hipHostAlloc(&M, A.len*B.len*2*sizeof(uint64), hipHostMallocMapped);
    uint64 *dev_M;
    hipHostGetDevicePointer(&dev_M, M, 0);
    bigInt* dev_pA;
    hipHostGetDevicePointer(&dev_pA, pA, 0);
    hipHostGetDevicePointer(&(*dev_pA).dev_data, A.data, 0);
    bigInt* dev_pB;
    hipHostGetDevicePointer(&dev_pB, pB, 0);
    hipHostGetDevicePointer(&(*dev_pB).dev_data, B.data, 0);

    kernal_create_table<<<1,A.len*B.len>>>(dev_M, dev_pA, dev_pB);
    hipDeviceSynchronize();

    // Sum Matrix
    dst.len = A.len+B.len;
    dst.data = (uint64*)calloc(8, dst.len);
    ///cudaHostAlloc(&(dst.data), dst.len<<3, cudaHostAllocMapped);
    uint64* carry = (uint64*)calloc(8, dst.len);

    //bigInt* dev_pDst;
    //cudaHostGetDevicePointer(&dev_pDst, pDst, 0);
    //cudaHostGetDevicePointer(&(*dev_pDst).dev_data, dst.data, 0);
    for (int i = 0; i < A.len; i++) {
        for (int j = 0; j < B.len; j++) {
            for (int k = 0; k < 2; k++) {
                carry[i+j+k+1] += has_carry(dst.data[i+j+k], *(M+i*(B.len*2)+(j*2)+k));
                dst.data[i+j+k] += *(M+(i*B.len*2)+(j*2)+k);
            }
        }
    }

    // Carry
    for (int i = 0; i < dst.len; i++) {
        carry[i+1] += has_carry(dst.data[i], carry[i]);
        dst.data[i] += carry[i];
    }

    // Trim zeros
    for (int i = dst.len-1; i >= 0; i--) {
        if (dst.data[i] != 0) {
            break;
        }
        dst.len -= 1;
    }
    dst.data = (uint64*)realloc(dst.data, (dst.len)<<3);

    *pDst = dst;
}

void show(uint64* num, int n) {
    for (int i = n-1; i >= 0; i--) {
        printf("%016lx ", num[i]);
    }
    printf("\n");
}


void test(bigInt* pA, bigInt* pB, bigInt* pDst_expected) {
    bigInt A = *pA;
    bigInt B = *pB;
    bigInt dst_expected = *pDst_expected;
    bigInt* pDst;
    hipHostAlloc(&pDst, sizeof(bigInt), hipHostMallocMapped);
    mul(pDst, pA, pB);
    bigInt dst = *pDst;
    if (VERBOSE) {
        show(A.data, A.len);
        show(B.data, B.len);
        show(dst_expected.data, dst_expected.len);
        show(dst.data, dst.len);
        printf("\n");
    } else {
        printf("OK\n");
    }
    assert(dst.len == dst_expected.len);
    for (int i = 0; i < dst.len; i++) {
        assert(dst.data[i] == dst_expected.data[i]);
    }
    return;
}

int main() {
    if (MAP_HOST) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        if (!prop.canMapHostMemory) {
            printf("Can not map host memory\n");
        }
        hipSetDeviceFlags(hipDeviceMapHost);
    }
    int fp = open("../testdata/small/numbers", O_RDONLY);
    int n;
    for (;;) {
        bigInt* nums[3];
        for (int i = 0; i < 3; i++) {
            hipHostAlloc(&nums[i], sizeof(bigInt), hipHostMallocMapped);
            if (read(fp, &n, sizeof(n)) == sizeof(n)) {
                hipHostAlloc(&((*nums[i]).data), ((n+7)/8)*sizeof(uint64), hipHostMallocMapped);
                if ((read(fp, (*nums[i]).data, n) != n)) {
                    assert(false);
                }
                (*nums[i]).len = (n+7)/8;
            } else {
                goto DONE;
            }
        }
        test(nums[0], nums[1], nums[2]);
    }
DONE:
    return 0;
}
