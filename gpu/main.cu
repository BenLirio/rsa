#include <fcntl.h>
#include <assert.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

typedef unsigned long int uint64;

typedef struct bigInt {
    uint64* data;
    int len;
} bigInt;

void panic(char* msg) {
    printf("%s\n", msg);
    exit(1);
}

uint64 mul_lo(uint64 A, uint64 B) {
    return A*B;
}

void mul_hi(uint64* dst, uint64 A, uint64 B) {
    __asm__(
            "umulh x1, x1, x2\n"
            "str x1, [x0]"
   );
}

bool has_carry(uint64 A, uint64 B) {
    bool rtn = ((A&B)|((A|B)&(~(A+B))))>>63 == 1;
    return rtn;
}

__global__ void kernal_create_table(uint64* M, uint64* A_data, uint64* B_data, int A_len, int B_len) {
    int i = threadIdx.x/B_len;
    int j = threadIdx.x%B_len;
    // cs.opensource.google/go/go/+/master:src/math/bits/bits.go
    // lo and hi multiplication for 64 bits
    uint64 x = A_data[i];
    uint64 y = B_data[j];
    uint64 x0 = (x<<32)>>32;
    uint64 x1 = x>>32;
    uint64 y0 = (y<<32)>>32;
    uint64 y1 = y>>32;
    uint64 w0 = x0*y0;
    uint64 t = x1*y0 + (w0>>32);
    uint64 w1 = (t<<32)>>32;
    uint64 w2 = t>>32;
    w1 += x0*y1;
    uint64 hi = x1*y1 + w2 + (w1>>32);
    uint64 lo = x*y;
    *(M+(i*B_len*2)+(j*2)+(1)) = hi;
    *(M+i*B_len*2+j*2) = lo;
}


void mul(bigInt* pDst, bigInt A, bigInt B) {
    bigInt dst = *pDst;
    // Create matrix
    uint64 M[A.len][B.len][2];
    uint64 *dev_M;
    size_t bytes = A.len*B.len*2*sizeof(uint64);
    hipMalloc(&dev_M, bytes);
    hipMemcpy(dev_M, M, bytes, hipMemcpyHostToDevice);
    uint64 *dev_A_data;
    hipMalloc(&dev_A_data, A.len*sizeof(uint64));
    hipMemcpy(dev_A_data, A.data, A.len*sizeof(uint64), hipMemcpyHostToDevice);
    uint64 *dev_B_data;
    hipMalloc(&dev_B_data, B.len*sizeof(uint64));
    hipMemcpy(dev_B_data, B.data, B.len*sizeof(uint64), hipMemcpyHostToDevice);
    kernal_create_table<<<1,A.len*B.len>>>(dev_M, dev_A_data, dev_B_data, A.len, B.len);

    hipMemcpy(M, dev_M, bytes, hipMemcpyDeviceToHost);


    // Sum Matrix
    dst.len = A.len+B.len;
    dst.data = (uint64*)calloc(8, dst.len);
    uint64* carry = (uint64*)calloc(8, dst.len);
    for (int i = 0; i < A.len; i++) {
        for (int j = 0; j < B.len; j++) {
            for (int k = 0; k < 2; k++) {
                carry[i+j+k+1] += has_carry(dst.data[i+j+k], M[i][j][k]);
                dst.data[i+j+k] += M[i][j][k];
            }
        }
    }

    // Carry
    for (int i = 0; i < dst.len; i++) {
        carry[i+1] += has_carry(dst.data[i], carry[i]);
        dst.data[i] += carry[i];
    }

    // Trim zeros
    for (int i = dst.len-1; i >= 0; i--) {
        if (dst.data[i] != 0) {
            break;
        }
        dst.len -= 1;
    }
    dst.data = (uint64*)realloc(dst.data, (dst.len)<<3);

    *pDst = dst;
}

void show(uint64* num, int n) {
    for (int i = n-1; i >= 0; i--) {
        printf("%016lx ", num[i]);
    }
    printf("\n");
}

bool VERBOSE = false;

void test(bigInt nums[]) {
    bigInt A = nums[0];
    bigInt B = nums[1];
    bigInt dst_expected = nums[2];
    bigInt dst;
    mul(&dst, A, B);
    if (VERBOSE) {
        for (int i = 0; i < 3; i++) {
            show(nums[i].data, nums[i].len);
        }
        show(dst.data, dst.len);
        printf("\n");
    } else {
        printf("OK\n");
    }
    assert(dst.len == dst_expected.len);
    for (int i = 0; i < dst.len; i++) {
        assert(dst.data[i] == dst_expected.data[i]);
    }
    
    return;
}

int main() {
    int fp = open("testdata/small/numbers", O_RDONLY);
    int n;
    for (;;) {
        bigInt nums[3];
        for (int i = 0; i < 3; i++) {
            if (read(fp, &n, sizeof(n)) == sizeof(n)) {
                nums[i].data = (uint64*)calloc(8, (n+7)/8);
                if ((read(fp, nums[i].data, n) != n)) {
                    assert(false);
                }
                nums[i].len = (n+7)/8;
            } else {
                goto DONE;
            }
        }
        test(nums);
    }
DONE:
    return 0;
}
